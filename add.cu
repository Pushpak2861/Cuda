
#include <hip/hip_runtime.h>
#include <iostream>
#define N 5
using namespace std;



__global__ void addEle(int *x , int *y , int *res){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    res[i] = x[i] + y[i];
}


int main(int argc , char **argv){
    int a[N] = {1,2,3,4,5};
    int b[N] = {6,7,8,9,10};
    int res[N];

    // create space in gpu
    int *d_x , *d_y , *d_res;
    hipMalloc(&d_x , N*sizeof(a[0]));
    hipMalloc(&d_y , N*sizeof(int));
    hipMalloc(&d_res , N*sizeof(int));

    hipMemcpy(d_x , a , N*sizeof(int) , hipMemcpyHostToDevice);
    hipMemcpy(d_y , b , N*sizeof(int) , hipMemcpyHostToDevice);

    addEle<<<1,N>>>(d_x , d_y , d_res);

    hipMemcpy(res,d_res ,N*sizeof(int) , hipMemcpyDeviceToHost);

    for (auto i : res){
        cout<< i << endl;
    }
}