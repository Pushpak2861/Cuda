
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>

using namespace std;
#define mask_len 7



__constant__ int Mask[mask_len];


__global__ void convolution(int* input , int* output , int N){

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int radius = mask_len / 2;
    int start = tid - radius;
    int temp = 0;
    for (int i =0 ; i< mask_len ; i++){
        if ((start + i >= 0 ) && (start + i < N)){
            temp += input[start+i] * Mask[i];

        }
    }

    output[tid] = temp;
}


void init_arr(int *a , int N){

    for (int i = 0 ; i<N ; i++){
        a[i] = rand() % 100;
    }
}

int main(int argc , char **argv){
    int N = 1 << 20;
    int bytes = N * sizeof(int);
    int bytes_m = mask_len * sizeof(int);
    int* out = new int[N];
    int* inp = new int[N];
    int mask[mask_len];
    int *input , *output;

    //allocate space in GPU

    hipMalloc(&input , bytes);
    hipMalloc(&output , bytes);

    init_arr(inp , N);
    init_arr(mask , mask_len);

    hipMemcpyToSymbol(HIP_SYMBOL(Mask) , mask , bytes_m);
    hipMemcpy(input , inp , bytes , hipMemcpyHostToDevice);


    int threads = 512;
    int blocks = (N + threads -1 )/threads;

    convolution<<< blocks , threads >>>(input , output , N);
    hipDeviceSynchronize();


    hipMemcpy(out , output , bytes , hipMemcpyDeviceToHost);
    

    for (int i=0; i<N; i++){
        cout << out[i] << endl;
    }

    delete[] out;
    delete[] inp;

}

