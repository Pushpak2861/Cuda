
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

#ifndef matrix
#define matrix
#endif 

#ifdef matrix
#define N 3
#endif


__global__ void matxmat(int *mat1 , int *mat2 , int *res){
    int row = threadIdx.y;
    int col = threadIdx.x;
    int sum1 = 0;

    for (int i =0 ; i<N; i++){
        sum1 += mat1[N * row + i] * mat2[i * N + col];
        
    }
    res[row * N +col] = sum1;
    
}

int main(int argc , char **argv){
    int mat1[N*N] = {1,2,3,4,5,6,7,8,9};
    int mat2[N*N] = {10,11,12,13,14,15,16,17,18};
    int res[N*N];

    int *d_mat1 , *d_mat2 , *d_res;

    hipMalloc(&d_mat1 , N*N*(sizeof(int)));
    hipMalloc(&d_mat2 , N*N*(sizeof(int)));
    hipMalloc(&d_res , N*N*(sizeof(int)));

    hipMemcpy(d_mat1 , mat1 , N*N*(sizeof(int)) , hipMemcpyHostToDevice);
    hipMemcpy(d_mat2 , mat2 , N*N*(sizeof(int)) , hipMemcpyHostToDevice);

    dim3 blockDim(3,3);
    matxmat<<<1 , blockDim>>>(d_mat1 , d_mat2 , d_res);

    hipMemcpy(res , d_res , N*N*sizeof(int) , hipMemcpyDeviceToHost);

    for (auto result : res){
        cout << result << endl;
    }

}