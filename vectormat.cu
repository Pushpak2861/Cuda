
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
using namespace std;

#ifndef flat
#define flat
#endif

#ifdef flat
#define N 3
#endif 


__global__ void vectoradd(int *mat , int *vec , int *res){
    int i = threadIdx.x;
    int sum1 = 0;
    for (int j =0 ; j<N ; j++){
        sum1 += mat[i*N + j] * vec[j];

    }

    res[i] = sum1;

}



int main(int argc , char **argv){
    int vec[N];
    int mat[N*N];
    int res[N];
    int sum1 = 1;
    int prevsum1 = sum1;
    for (int i=0 ; i<N ; i++){
        vec[i] = i;

    }

    for (int i=0; i<N*N ; i++){
        mat[i] = sum1;
        sum1 = sum1 + prevsum1;
        prevsum1 = mat[i];
    }

    int *d_vec , *d_mat; // *d_vec is a pointer in host(cpu) whose address in cpu itself
    int*d_res;
    // a pointer has two things 1) its own address 2) address which its pointing

    hipMalloc(&d_mat ,N*N*sizeof(int)); // now we give the address of the pointer to store the address of allocated space in gpu
    hipMalloc(&d_vec ,N*sizeof(int));
    hipMalloc(&d_res , N*sizeof(int));

    hipMemcpy(d_mat , mat , N*N*sizeof(int) , hipMemcpyHostToDevice);
    hipMemcpy(d_vec , vec , N*sizeof(int) , hipMemcpyHostToDevice);

    vectoradd<<<1,N>>>(d_mat , d_vec , d_res);

    hipMemcpy(res , d_res , N*sizeof(int) , hipMemcpyDeviceToHost);

    for (auto i : res){
        cout<< i << endl;
    }

}
